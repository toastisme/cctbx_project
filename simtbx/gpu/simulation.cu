#include "hip/hip_runtime.h"
#include <scitbx/array_family/boost_python/flex_fwd.h>
#include <cudatbx/cuda_base.cuh>
#include <simtbx/gpu/simulation.h>
#include <simtbx/gpu/simulation.cuh>
#include <scitbx/array_family/flex_types.h>
#define THREADS_PER_BLOCK_X 128
#define THREADS_PER_BLOCK_Y 1
#define THREADS_PER_BLOCK_TOTAL (THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y)

namespace simtbx {
namespace gpu {

namespace af = scitbx::af;
//refactor later into helper file
  static hipError_t cudaMemcpyVectorDoubleToDevice(CUDAREAL *dst, const double *src, size_t vector_items) {
        CUDAREAL * temp = new CUDAREAL[vector_items];
        for (size_t i = 0; i < vector_items; i++) {
                temp[i] = src[i];
        }
        hipError_t ret = hipMemcpy(dst, temp, sizeof(*dst) * vector_items, hipMemcpyHostToDevice);
        delete temp;
        return ret;
  }

/* make a unit vector pointing in same direction and report magnitude (both args can be same vector) */
  double cpu_unitize(const double * vector, double * new_unit_vector) {

        double v1 = vector[1];
        double v2 = vector[2];
        double v3 = vector[3];

        double mag = sqrt(v1 * v1 + v2 * v2 + v3 * v3);

        if (mag != 0.0) {
                /* normalize it */
                new_unit_vector[0] = mag;
                new_unit_vector[1] = v1 / mag;
                new_unit_vector[2] = v2 / mag;
                new_unit_vector[3] = v3 / mag;
        } else {
                /* can't normalize, report zero vector */
                new_unit_vector[0] = 0.0;
                new_unit_vector[1] = 0.0;
                new_unit_vector[2] = 0.0;
                new_unit_vector[3] = 0.0;
        }
        return mag;
  }

  void
  exascale_api::show(){
    SCITBX_EXAMINE(SIM.roi_xmin);
    SCITBX_EXAMINE(SIM.roi_xmax);
    SCITBX_EXAMINE(SIM.roi_ymin);
    SCITBX_EXAMINE(SIM.roi_ymax);
    SCITBX_EXAMINE(SIM.oversample);
    SCITBX_EXAMINE(SIM.point_pixel);
    SCITBX_EXAMINE(SIM.pixel_size);
    SCITBX_EXAMINE(cu_subpixel_size);
    SCITBX_EXAMINE(cu_steps);
    SCITBX_EXAMINE(SIM.detector_thickstep);
    SCITBX_EXAMINE(SIM.detector_thicksteps);
    SCITBX_EXAMINE(SIM.detector_thick);
    SCITBX_EXAMINE(SIM.detector_attnlen);
    SCITBX_EXAMINE(SIM.curved_detector);
    SCITBX_EXAMINE(SIM.distance);
    SCITBX_EXAMINE(SIM.close_distance);
    SCITBX_EXAMINE(SIM.dmin);
    SCITBX_EXAMINE(SIM.phi0);
    SCITBX_EXAMINE(SIM.phistep);
    SCITBX_EXAMINE(SIM.phisteps);
    SCITBX_EXAMINE(SIM.sources);
    SCITBX_EXAMINE(SIM.mosaic_spread);
    SCITBX_EXAMINE(SIM.mosaic_domains);
    SCITBX_EXAMINE(SIM.Na);
    SCITBX_EXAMINE(SIM.Nb);
    SCITBX_EXAMINE(SIM.Nc);
    SCITBX_EXAMINE(SIM.fluence);
    SCITBX_EXAMINE(SIM.spot_scale);
    SCITBX_EXAMINE(SIM.integral_form);
    SCITBX_EXAMINE(SIM.default_F);
    SCITBX_EXAMINE(SIM.interpolate);
    SCITBX_EXAMINE(SIM.nopolar);
    SCITBX_EXAMINE(SIM.polarization);
    SCITBX_EXAMINE(SIM.fudge);
  }

  void
  exascale_api::add_energy_channel_from_gpu_amplitudes_cuda(
    int const& ichannel,
    simtbx::gpu::gpu_energy_channels & gec,
    simtbx::gpu::gpu_detector & gdt
  ){
        cudaSafeCall(hipSetDevice(SIM.device_Id));

        // transfer source_I, source_lambda
        // the int arguments are for sizes of the arrays
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_I, SIM.source_I, SIM.sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_lambda, SIM.source_lambda, SIM.sources));

        // magic happens here: take pointer from singleton, temporarily use it for add Bragg iteration:
        cu_current_channel_Fhkl = gec.d_channel_Fhkl[ichannel];

        hipDeviceProp_t deviceProps = { 0 };
        cudaSafeCall(hipGetDeviceProperties(&deviceProps, SIM.device_Id));
        int smCount = deviceProps.multiProcessorCount;
        dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
        dim3 numBlocks(smCount * 8, 1);

        std::size_t panel_size = gdt.cu_slow_pixels * gdt.cu_fast_pixels;
        const int vec_len = 4;

        // the for loop around panels.  Offsets given.
        for (std::size_t idx_p = 0; idx_p < gdt.cu_n_panels; idx_p++){
          // loop thru panels and increment the array ptrs
          nanoBraggSpotsCUDAKernel<<<numBlocks, threadsPerBlock>>>(
          gdt.cu_slow_pixels, gdt.cu_fast_pixels, SIM.roi_xmin,
          SIM.roi_xmax, SIM.roi_ymin, SIM.roi_ymax, SIM.oversample, SIM.point_pixel,
          SIM.pixel_size, cu_subpixel_size, cu_steps, SIM.detector_thickstep, SIM.detector_thicksteps,
          SIM.detector_thick, SIM.detector_attnlen,
          &(gdt.cu_sdet_vector[vec_len * idx_p]),
          &(gdt.cu_fdet_vector[vec_len * idx_p]),
          &(gdt.cu_odet_vector[vec_len * idx_p]),
          &(gdt.cu_pix0_vector[vec_len * idx_p]),
          SIM.curved_detector, gdt.metrology.dists[idx_p], gdt.metrology.dists[idx_p], cu_beam_vector,
          gdt.metrology.Xbeam[idx_p], gdt.metrology.Ybeam[idx_p],
          SIM.dmin, SIM.phi0, SIM.phistep, SIM.phisteps, cu_spindle_vector,
          SIM.sources, cu_source_X, cu_source_Y, cu_source_Z,
          cu_source_I, cu_source_lambda, cu_a0, cu_b0,
          cu_c0, SIM.xtal_shape, SIM.mosaic_spread, SIM.mosaic_domains, cu_mosaic_umats,
          SIM.Na, SIM.Nb, SIM.Nc, SIM.V_cell,
          cu_water_size, cu_water_F, cu_water_MW, simtbx::nanoBragg::r_e_sqr, SIM.fluence,
          simtbx::nanoBragg::Avogadro, SIM.spot_scale, SIM.integral_form, SIM.default_F,
          SIM.interpolate, cu_current_channel_Fhkl, gec.cu_FhklParams, SIM.nopolar,
          cu_polar_vector, SIM.polarization, SIM.fudge,
          /* &(gdt.cu_maskimage[panel_size * idx_p]), */
          NULL,
          &(gdt.cu_floatimage[panel_size * idx_p]) /*out*/,
          &(gdt.cu_omega_reduction[panel_size * idx_p]) /*out*/,
          &(gdt.cu_max_I_x_reduction[panel_size * idx_p]) /*out*/,
          &(gdt.cu_max_I_y_reduction[panel_size * idx_p]) /*out*/,
          &(gdt.cu_rangemap[panel_size * idx_p]) /*out*/);

          cudaSafeCall(hipPeekAtLastError());
        }
        cudaSafeCall(hipDeviceSynchronize());

        //don't want to free the gec data when the nanoBragg goes out of scope, so switch the pointer
        cu_current_channel_Fhkl = NULL;

        add_array_CUDAKernel<<<numBlocks, threadsPerBlock>>>(gdt.cu_accumulate_floatimage,
          gdt.cu_floatimage,
          gdt.cu_n_panels * gdt.cu_slow_pixels * gdt.cu_fast_pixels);
  }

  void
  exascale_api::add_energy_channel_mask_allpanel_cuda(
    int const& ichannel,
    simtbx::gpu::gpu_energy_channels & gec,
    simtbx::gpu::gpu_detector & gdt,
    af::shared<bool> all_panel_mask
  ){
        cudaSafeCall(hipSetDevice(SIM.device_Id));

        // here or there, need to convert the all_panel_mask (3D map) into a 1D list of accepted pixels
        // coordinates for the active pixel list are absolute offsets into the detector array
        af::shared<int> active_pixel_list;
        const bool* jptr = all_panel_mask.begin();
        for (int j=0; j < all_panel_mask.size(); ++j){
          if (jptr[j]) {
            active_pixel_list.push_back(j);
          }
        }
        gdt.set_active_pixels_on_GPU(active_pixel_list);

        // transfer source_I, source_lambda
        // the int arguments are for sizes of the arrays
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_I, SIM.source_I, SIM.sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_lambda, SIM.source_lambda, SIM.sources));

        // magic happens here: take pointer from singleton, temporarily use it for add Bragg iteration:
        cu_current_channel_Fhkl = gec.d_channel_Fhkl[ichannel];

        hipDeviceProp_t deviceProps = { 0 };
        cudaSafeCall(hipGetDeviceProperties(&deviceProps, SIM.device_Id));
        int smCount = deviceProps.multiProcessorCount;
        dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
        dim3 numBlocks(smCount * 8, 1);

        const int vec_len = 4;
        // for call for all panels at the same time

          debranch_maskall_CUDAKernel<<<numBlocks, threadsPerBlock>>>(
          gdt.cu_n_panels, gdt.cu_slow_pixels, gdt.cu_fast_pixels, active_pixel_list.size(),
          SIM.oversample, SIM.point_pixel,
          SIM.pixel_size, cu_subpixel_size, cu_steps,
          SIM.detector_thickstep, SIM.detector_thicksteps,
          SIM.detector_thick, SIM.detector_attnlen,
          vec_len,
          gdt.cu_sdet_vector,
          gdt.cu_fdet_vector,
          gdt.cu_odet_vector,
          gdt.cu_pix0_vector,
          gdt.cu_distance, gdt.cu_distance, cu_beam_vector,
          gdt.cu_Xbeam, gdt.cu_Ybeam,
          SIM.dmin, SIM.phi0, SIM.phistep, SIM.phisteps, cu_spindle_vector,
          SIM.sources, cu_source_X, cu_source_Y, cu_source_Z,
          cu_source_I, cu_source_lambda, cu_a0, cu_b0,
          cu_c0, SIM.xtal_shape, SIM.mosaic_domains, cu_mosaic_umats,
          SIM.Na, SIM.Nb, SIM.Nc, SIM.V_cell,
          cu_water_size, cu_water_F, cu_water_MW, simtbx::nanoBragg::r_e_sqr, SIM.fluence,
          simtbx::nanoBragg::Avogadro, SIM.spot_scale, SIM.integral_form, SIM.default_F,
          cu_current_channel_Fhkl, gec.cu_FhklParams, SIM.nopolar,
          cu_polar_vector, SIM.polarization, SIM.fudge,
          gdt.cu_active_pixel_list,
          gdt.cu_floatimage /*out*/,
          gdt.cu_omega_reduction /*out*/,
          gdt.cu_max_I_x_reduction /*out*/,
          gdt.cu_max_I_y_reduction /*out*/,
          gdt.cu_rangemap /*out*/);

          cudaSafeCall(hipPeekAtLastError());
        cudaSafeCall(hipDeviceSynchronize());

        //don't want to free the gec data when the nanoBragg goes out of scope, so switch the pointer
        cu_current_channel_Fhkl = NULL;

        add_array_CUDAKernel<<<numBlocks, threadsPerBlock>>>(gdt.cu_accumulate_floatimage,
          gdt.cu_floatimage,
          gdt.cu_n_panels * gdt.cu_slow_pixels * gdt.cu_fast_pixels);
  }


  void
  exascale_api::add_background_cuda(simtbx::gpu::gpu_detector & gdt){
        cudaSafeCall(hipSetDevice(SIM.device_Id));

        // transfer source_I, source_lambda
        // the int arguments are for sizes of the arrays
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_I, SIM.source_I, SIM.sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_lambda, SIM.source_lambda, SIM.sources));

        CUDAREAL * cu_stol_of;
        cudaSafeCall(hipMalloc((void ** )&cu_stol_of, sizeof(*cu_stol_of) * SIM.stols));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_stol_of, SIM.stol_of, SIM.stols));

        CUDAREAL * cu_Fbg_of;
        cudaSafeCall(hipMalloc((void ** )&cu_Fbg_of, sizeof(*cu_Fbg_of) * SIM.stols));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_Fbg_of, SIM.Fbg_of, SIM.stols));

        hipDeviceProp_t deviceProps = { 0 };
        cudaSafeCall(hipGetDeviceProperties(&deviceProps, SIM.device_Id));
        int smCount = deviceProps.multiProcessorCount;
        dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
        dim3 numBlocks(smCount * 8, 1);

        //  initialize the device memory within a kernel.
        //  modify the arguments to initialize multipanel detector.
        nanoBraggSpotsInitCUDAKernel<<<numBlocks, threadsPerBlock>>>(
          gdt.cu_n_panels * gdt.cu_slow_pixels, gdt.cu_fast_pixels,
          gdt.cu_floatimage, gdt.cu_omega_reduction,
          gdt.cu_max_I_x_reduction, gdt.cu_max_I_y_reduction,
          gdt.cu_rangemap);
        cudaSafeCall(hipPeekAtLastError());
        cudaSafeCall(hipDeviceSynchronize());

        std::size_t panel_size = gdt.cu_slow_pixels * gdt.cu_fast_pixels;
        const int vec_len = 4;

        // the for loop around panels.  Offsets given.
        for (std::size_t idx_p = 0; idx_p < gdt.cu_n_panels; idx_p++){
          add_background_CUDAKernel<<<numBlocks, threadsPerBlock>>>(SIM.sources,
          SIM.oversample,
          SIM.pixel_size, gdt.cu_slow_pixels, gdt.cu_fast_pixels, SIM.detector_thicksteps,
          SIM.detector_thickstep, SIM.detector_attnlen,
          &(gdt.cu_sdet_vector[vec_len * idx_p]),
          &(gdt.cu_fdet_vector[vec_len * idx_p]),
          &(gdt.cu_odet_vector[vec_len * idx_p]),
          &(gdt.cu_pix0_vector[vec_len * idx_p]),
          gdt.metrology.dists[idx_p], SIM.point_pixel, SIM.detector_thick,
          cu_source_X, cu_source_Y, cu_source_Z,
          cu_source_lambda, cu_source_I,
          SIM.stols, cu_stol_of, cu_Fbg_of,
          SIM.nopolar, SIM.polarization, cu_polar_vector,
          simtbx::nanoBragg::r_e_sqr, SIM.fluence, SIM.amorphous_molecules,
          &(gdt.cu_floatimage[panel_size * idx_p]) /*out*/);

          cudaSafeCall(hipPeekAtLastError());
        }
        cudaSafeCall(hipDeviceSynchronize());
        add_array_CUDAKernel<<<numBlocks, threadsPerBlock>>>(gdt.cu_accumulate_floatimage,
          gdt.cu_floatimage,
          gdt.cu_n_panels * gdt.cu_slow_pixels * gdt.cu_fast_pixels);

        cudaSafeCall(hipFree(cu_stol_of));
        cudaSafeCall(hipFree(cu_Fbg_of));
}

  void
  exascale_api::allocate_cuda(){
    cudaSafeCall(hipSetDevice(SIM.device_Id));

    /* water_size not defined in class, CLI argument, defaults to 0 */
    double water_size = 0.0;
    /* missing constants */
    double water_F = 2.57;
    double water_MW = 18.0;

    /* make sure we are normalizing with the right number of sub-steps */
    int nb_steps = SIM.phisteps*SIM.mosaic_domains*SIM.oversample*SIM.oversample;
    double nb_subpixel_size = SIM.pixel_size/SIM.oversample;

        /*create transfer arguments to device space*/
        cu_subpixel_size = nb_subpixel_size; //check for conflict?
        cu_steps = nb_steps; //check for conflict?

        /* presumably thickness and attenuation can be migrated to the gpu detector class XXX FIXME*/
        //cu_detector_thick = SIM.detector_thick;
        //cu_detector_mu = SIM.detector_attnlen; // synonyms
        //cu_distance = SIM.distance; /* distance and close distance, detector properties? XXX FIXME */
        //cu_close_distance = SIM.close_distance;

        cu_water_size = water_size;
        cu_water_F = water_F;
        cu_water_MW = water_MW;

        const int vector_length = 4;
        int cu_sources = SIM.sources;
        int cu_mosaic_domains = SIM.mosaic_domains;

        cudaSafeCall(hipMalloc((void ** )&cu_beam_vector, sizeof(*cu_beam_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_beam_vector, SIM.beam_vector, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_spindle_vector, sizeof(*cu_spindle_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_spindle_vector, SIM.spindle_vector, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_a0, sizeof(*cu_a0) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_a0, SIM.a0, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_b0, sizeof(*cu_b0) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_b0, SIM.b0, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_c0, sizeof(*cu_c0) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_c0, SIM.c0, vector_length));

        // Unitize polar vector before sending it to the GPU.
        // Optimization do it only once here rather than multiple time per pixel in the GPU.
        double polar_vector_unitized[4];
        cpu_unitize(SIM.polar_vector, polar_vector_unitized);
        cudaSafeCall(hipMalloc((void ** )&cu_polar_vector, sizeof(*cu_polar_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_polar_vector, polar_vector_unitized, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_source_X, sizeof(*cu_source_X) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_X, SIM.source_X, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_source_Y, sizeof(*cu_source_Y) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_Y, SIM.source_Y, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_source_Z, sizeof(*cu_source_Z) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_Z, SIM.source_Z, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_source_I, sizeof(*cu_source_I) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_I, SIM.source_I, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_source_lambda, sizeof(*cu_source_lambda) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_lambda, SIM.source_lambda, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_mosaic_umats, sizeof(*cu_mosaic_umats) * cu_mosaic_domains * 9));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_mosaic_umats, SIM.mosaic_umats, cu_mosaic_domains * 9));
  };

  exascale_api::~exascale_api(){
    cudaSafeCall(hipSetDevice(SIM.device_Id));

        cudaSafeCall(hipFree(cu_beam_vector));
        cudaSafeCall(hipFree(cu_spindle_vector));
        cudaSafeCall(hipFree(cu_source_X));
        cudaSafeCall(hipFree(cu_source_Y));
        cudaSafeCall(hipFree(cu_source_Z));
        cudaSafeCall(hipFree(cu_source_I));
        cudaSafeCall(hipFree(cu_source_lambda));
        cudaSafeCall(hipFree(cu_a0));
        cudaSafeCall(hipFree(cu_b0));
        cudaSafeCall(hipFree(cu_c0));
        cudaSafeCall(hipFree(cu_mosaic_umats));
        cudaSafeCall(hipFree(cu_polar_vector));
  }

} // gpu
} // simtbx
