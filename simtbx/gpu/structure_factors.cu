#include <cudatbx/cuda_base.cuh>
#include <simtbx/gpu/structure_factors.h>

namespace simtbx {
namespace gpu {

  gpu_energy_channels::gpu_energy_channels(int const& deviceId){
    h_deviceID = deviceId;
    hipSetDevice(deviceId);
  }

  void
  gpu_energy_channels::structure_factors_to_GPU_detail(af::shared<double> linear_amplitudes){
    double * raw_ptr = linear_amplitudes.begin();
    CUDAREAL * cu_Fhkl = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_Fhkl,
                     sizeof(*cu_Fhkl) * linear_amplitudes.size()));
    cudaSafeCall(hipMemcpy(cu_Fhkl, raw_ptr,
                     sizeof(*cu_Fhkl) * linear_amplitudes.size(), hipMemcpyHostToDevice));

    d_channel_Fhkl.push_back(cu_Fhkl);

    if (d_channel_Fhkl.size()==1) { //first time through send ranges to device
      hklParams FhklParams = { h_range * k_range * l_range,
                             h_min, h_max, h_range, k_min, k_max, k_range, l_min, l_max, l_range };

      cudaSafeCall(hipMalloc((void ** )&cu_FhklParams, sizeof(*cu_FhklParams)));
      cudaSafeCall(hipMemcpy(cu_FhklParams, &FhklParams, sizeof(*cu_FhklParams), hipMemcpyHostToDevice));
    }
  }

  void gpu_energy_channels::free_detail(){
    cudaSafeCall(hipSetDevice(h_deviceID));
    for (int i_cu_ptr=0; i_cu_ptr < d_channel_Fhkl.size(); ++i_cu_ptr){
      cudaSafeCall(hipFree(d_channel_Fhkl[i_cu_ptr]));
    }
    cudaSafeCall(hipFree(cu_FhklParams));
  }
} // gpu
} // simtbx
