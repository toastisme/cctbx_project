#include <scitbx/array_family/boost_python/flex_fwd.h>
#include <cudatbx/cuda_base.cuh>
#include <simtbx/gpu/detector.h>
#include <simtbx/gpu/detector.cuh>
#include <scitbx/vec3.h>
#include <scitbx/vec2.h>
#define THREADS_PER_BLOCK_X 128
#define THREADS_PER_BLOCK_Y 1
#define THREADS_PER_BLOCK_TOTAL (THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y)

namespace simtbx {
namespace gpu {

//refactor later into helper file
  static hipError_t detMemcpyVectorDoubleToDevice(CUDAREAL *dst, const double *src, size_t vector_items) {
        CUDAREAL * temp = new CUDAREAL[vector_items];
        for (size_t i = 0; i < vector_items; i++) {
                temp[i] = src[i];
        }
        hipError_t ret = hipMemcpy(dst, temp, sizeof(*dst) * vector_items, hipMemcpyHostToDevice);
        delete temp;
        return ret;
  }

  packed_metrology::packed_metrology(dxtbx::model::Detector const & arg_detector,
                                   dxtbx::model::MonoBeam const & arg_beam) {

    for (std::size_t panel_id = 0; panel_id < arg_detector.size(); panel_id++){
          // helper code arising from the nanoBragg constructor, with user_beam=True
      typedef scitbx::vec3<double> vec3;

      /* DETECTOR properties */
      /* typically: 1 0 0 */
      vec3 fdet_vector = arg_detector[panel_id].get_fast_axis();
      fdet_vector = fdet_vector.normalize();

      /* typically: 0 -1 0 */
      vec3 sdet_vector = arg_detector[panel_id].get_slow_axis();
      sdet_vector = sdet_vector.normalize();

      /* set orthogonal vector to the detector pixel array */
      vec3 odet_vector = fdet_vector.cross(sdet_vector);
      odet_vector = odet_vector.normalize();

      /* dxtbx origin is location of outer corner of the first pixel */
      vec3 pix0_vector = arg_detector[panel_id].get_origin()/1000.0;

      /* what is the point of closest approach between sample and detector? */
      double close_distance = pix0_vector * odet_vector;
      if (close_distance < 0){
        bool verbose = false;
        if(verbose)printf("WARNING: dxtbx model is lefthanded. Inverting odet_vector.\n");
        odet_vector = -1. * odet_vector;
        close_distance = -1*close_distance;
      }

      sdet.push_back(sdet_vector.length());
      fdet.push_back(fdet_vector.length());
      odet.push_back(odet_vector.length());
      pix0.push_back(0.);
      for (std::size_t idx_vec = 0; idx_vec < 3; idx_vec++){
            sdet.push_back(sdet_vector[idx_vec]);
            fdet.push_back(fdet_vector[idx_vec]);
            odet.push_back(odet_vector[idx_vec]);
            pix0.push_back(pix0_vector[idx_vec]);
      }
      /* set beam centre */
      scitbx::vec2<double> dials_bc=arg_detector[panel_id].get_beam_centre(arg_beam.get_s0());
      dists.push_back(close_distance);
      Xbeam.push_back(dials_bc[0]/1000.0);
      Ybeam.push_back(dials_bc[1]/1000.0);
    }
  };

  packed_metrology::packed_metrology(const simtbx::nanoBragg::nanoBragg& nB){
      for (std::size_t idx_vec = 0; idx_vec < 4; idx_vec++){
            sdet.push_back(nB.sdet_vector[idx_vec]);
            fdet.push_back(nB.fdet_vector[idx_vec]);
            odet.push_back(nB.odet_vector[idx_vec]);
            pix0.push_back(nB.pix0_vector[idx_vec]);
      }
      dists.push_back(nB.close_distance);
      Xbeam.push_back(nB.Xbeam);
      Ybeam.push_back(nB.Ybeam);
  }

  void
  packed_metrology::show() const {
    for (std::size_t idx_p = 0; idx_p < Xbeam.size(); idx_p++){
      printf(" Panel %3d\n",idx_p);
      printf(" Panel %3d sdet %9.6f %9.6f %9.6f %9.6f fdet %9.6f %9.6f %9.6f %9.6f\n",
             idx_p,sdet[4*idx_p+0],sdet[4*idx_p+1],sdet[4*idx_p+2],sdet[4*idx_p+3],
                          fdet[4*idx_p+0],fdet[4*idx_p+1],fdet[4*idx_p+2],fdet[4*idx_p+3]
      );
      printf(" Panel %3d odet %9.6f %9.6f %9.6f %9.6f pix0 %9.6f %9.6f %9.6f %9.6f\n",
             idx_p,odet[4*idx_p+0],odet[4*idx_p+1],odet[4*idx_p+2],odet[4*idx_p+3],
                          pix0[4*idx_p+0],pix0[4*idx_p+1],pix0[4*idx_p+2],pix0[4*idx_p+3]
      );
      printf(" Panel %3d beam %11.8f %11.8f\n",idx_p,Xbeam[idx_p],Ybeam[idx_p]);
    }
  }

  void
  gpu_detector::construct_detail(int const& arg_device_id,
                                 dxtbx::model::Detector const & arg_detector){
    hipSetDevice(arg_device_id);

    //1) determine the size
    cu_n_panels = detector.size();
    SCITBX_ASSERT( cu_n_panels >= 1);

    //2) confirm that array dimensions are similar for each size
    cu_slow_pixels = detector[0].get_image_size()[1];
    cu_fast_pixels = detector[0].get_image_size()[0];
    for (int ipanel=1; ipanel < detector.size(); ++ipanel){
      SCITBX_ASSERT(detector[ipanel].get_image_size()[1] == cu_slow_pixels);
      SCITBX_ASSERT(detector[ipanel].get_image_size()[0] == cu_fast_pixels);
    }
    _image_size = cu_n_panels * cu_slow_pixels * cu_fast_pixels;

    //3) allocate a cuda array with these dimensions
    /* separate accumulator image outside the usual nanoBragg data structure.
           1. accumulate contributions from a sequence of source energy channels computed separately
           2. represent multiple panels, all same rectangular shape; slowest dimension = n_panels */
    cudaSafeCall(hipMalloc((void ** )&cu_accumulate_floatimage,
                            sizeof(*cu_accumulate_floatimage) * _image_size));
    cudaSafeCall(hipMemset((void *)cu_accumulate_floatimage, 0,
                            sizeof(*cu_accumulate_floatimage) * _image_size));
  };

  gpu_detector::gpu_detector(int const& arg_device_id,
                             dxtbx::model::Detector const & arg_detector,
                             dxtbx::model::MonoBeam const& arg_beam):
    h_deviceID(arg_device_id),
    detector(arg_detector),
    cu_active_pixel_list(NULL),
    cu_accumulate_floatimage(NULL),
    metrology(arg_detector, arg_beam){
    construct_detail(arg_device_id, arg_detector);
  }

  gpu_detector::gpu_detector(int const& arg_device_id,
                             const simtbx::nanoBragg::nanoBragg& nB):
    h_deviceID(arg_device_id),
    metrology(nB),
    cu_active_pixel_list(NULL),
    cu_accumulate_floatimage(NULL){
    hipSetDevice(arg_device_id);

    //1) determine the size
    cu_n_panels = 1;

    //2) confirm that array dimensions are similar for each size
    cu_slow_pixels = nB.spixels;
    cu_fast_pixels = nB.fpixels;
    _image_size = cu_n_panels * cu_slow_pixels * cu_fast_pixels;

    //3) allocate a cuda array with these dimensions
    /* separate accumulator image outside the usual nanoBragg data structure.
           1. accumulate contributions from a sequence of source energy channels computed separately
           2. represent multiple panels, all same rectangular shape; slowest dimension = n_panels */
    cudaSafeCall(hipMalloc((void ** )&cu_accumulate_floatimage,
                            sizeof(*cu_accumulate_floatimage) * _image_size));
    cudaSafeCall(hipMemset((void *)cu_accumulate_floatimage, 0,
                            sizeof(*cu_accumulate_floatimage) * _image_size));
  }

  void gpu_detector::free_detail(){
    hipSetDevice(h_deviceID);
    //4) make sure we can deallocate cuda array later on
    if (cu_accumulate_floatimage != NULL) {
      cudaSafeCall(hipFree(cu_accumulate_floatimage));
    }
  };

  void
  gpu_detector::scale_in_place(const double& factor){
    cudaSafeCall(hipSetDevice(h_deviceID));
    hipDeviceProp_t deviceProps = { 0 };
    cudaSafeCall(hipGetDeviceProperties(&deviceProps, h_deviceID));
  int smCount = deviceProps.multiProcessorCount;
  dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
  dim3 numBlocks(smCount * 8, 1);
  int total_pixels = _image_size;
  scale_array_CUDAKernel<<<numBlocks, threadsPerBlock>>>(
    factor, cu_accumulate_floatimage, total_pixels);
  }

  void
  gpu_detector::write_raw_pixels(simtbx::nanoBragg::nanoBragg& nB){
    //only implement the monolithic detector case, one panel
    SCITBX_ASSERT(nB.spixels == cu_slow_pixels);
    SCITBX_ASSERT(nB.fpixels == cu_fast_pixels);
    SCITBX_ASSERT(cu_n_panels == 1);
    /* nB.raw_pixels = af::flex_double(af::flex_grid<>(nB.spixels,nB.fpixels));
       do not reallocate CPU memory for the data write, as it is not needed
     */
    double * double_floatimage = nB.raw_pixels.begin();
    cudaSafeCall(hipSetDevice(nB.device_Id));
    cudaSafeCall(hipMemcpy(
     double_floatimage,
     cu_accumulate_floatimage,
     sizeof(*cu_accumulate_floatimage) * _image_size,
     hipMemcpyDeviceToHost));
  }

  af::flex_double
  gpu_detector::get_raw_pixels(){
    //return the data array for the multipanel detector case
    af::flex_double z(af::flex_grid<>(cu_n_panels,cu_slow_pixels,cu_fast_pixels), af::init_functor_null<double>());
    double* begin = z.begin();
    cudaSafeCall(hipSetDevice(h_deviceID));
    cudaSafeCall(hipMemcpy(
     begin,
     cu_accumulate_floatimage,
     sizeof(*cu_accumulate_floatimage) * _image_size,
     hipMemcpyDeviceToHost));
    return z;
  }

  void
  gpu_detector::set_active_pixels_on_GPU(af::shared<std::size_t> active_pixel_list_value){
    active_pixel_list = active_pixel_list_value;
    cudaSafeCall(hipSetDevice(h_deviceID));
    std::size_t * ptr_active_pixel_list = active_pixel_list.begin();
    cudaSafeCall(hipMalloc((void ** )&cu_active_pixel_list, sizeof(*cu_active_pixel_list) * active_pixel_list.size() ));
    cudaSafeCall(hipMemcpy(cu_active_pixel_list,
                            ptr_active_pixel_list,
                            sizeof(*cu_active_pixel_list) * active_pixel_list.size(),
                            hipMemcpyHostToDevice));
  }

  af::shared<double>
  gpu_detector::get_whitelist_raw_pixels(af::shared<std::size_t> selection
  ){
    //return the data array for the multipanel detector case, but only for whitelist pixels
    af::shared<double> z(selection.size(), af::init_functor_null<double>());
    double* begin = z.begin();
    cudaSafeCall(hipSetDevice(h_deviceID));
    CUDAREAL * cu_active_pixel_results;
    std::size_t * cu_active_pixel_selection;

    cudaSafeCall(hipMalloc((void ** )&cu_active_pixel_results, sizeof(*cu_active_pixel_results) * z.size() ));
    cudaSafeCall(hipMalloc((void ** )&cu_active_pixel_selection, sizeof(*cu_active_pixel_selection) * z.size() ));
    cudaSafeCall(hipMemcpy(cu_active_pixel_selection,
                 selection.begin(), sizeof(*cu_active_pixel_selection) * z.size(),
                 hipMemcpyHostToDevice));

    hipDeviceProp_t deviceProps = { 0 };
    cudaSafeCall(hipGetDeviceProperties(&deviceProps, h_deviceID));
    int smCount = deviceProps.multiProcessorCount;
    dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
    dim3 numBlocks(smCount * 8, 1);
    int total_pixels = z.size();
    get_active_pixel_selection_CUDAKernel<<<numBlocks, threadsPerBlock>>>(
      cu_active_pixel_results, cu_active_pixel_selection, cu_accumulate_floatimage, z.size());

    cudaSafeCall(hipMemcpy(
      begin,
      cu_active_pixel_results,
      sizeof(*cu_active_pixel_results) * z.size(),
      hipMemcpyDeviceToHost));
    cudaSafeCall(hipFree(cu_active_pixel_selection));
    cudaSafeCall(hipFree(cu_active_pixel_results));
    return z;
  }

  void
  gpu_detector::each_image_allocate(){
    hipSetDevice(h_deviceID);
    /*allocate but do not initialize (set to 0) the reductions (the code was too inefficient and was removed as the reductions
      are not utilized in practice.  Should they be needed in the future a faster zeroing API must be found*/
    cu_omega_reduction = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_omega_reduction, sizeof(*cu_omega_reduction) * _image_size));

    cu_max_I_x_reduction = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_max_I_x_reduction, sizeof(*cu_max_I_x_reduction) * _image_size));

    cu_max_I_y_reduction = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_max_I_y_reduction, sizeof(*cu_max_I_y_reduction) * _image_size));

    cu_rangemap = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_rangemap, sizeof(*cu_rangemap) * _image_size));

    cu_maskimage = NULL;
    int unsigned short * maskimage = NULL; //default case, must implement non-trivial initializer elsewhere
    if (maskimage != NULL) {
      cudaSafeCall(hipMalloc((void ** )&cu_maskimage, sizeof(*cu_maskimage) * _image_size));
      cudaSafeCall(hipMemcpy(cu_maskimage, maskimage, sizeof(*cu_maskimage) * _image_size,
                   hipMemcpyHostToDevice));
    }

    // In contrast to old API, new API initializes its own accumulator, does not take values from CPU
    cu_floatimage = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_floatimage, sizeof(*cu_floatimage) * _image_size));

        const int met_length = metrology.sdet.size();
        cudaSafeCall(hipMalloc((void ** )&cu_sdet_vector, sizeof(*cu_sdet_vector) * met_length));
        cudaSafeCall(detMemcpyVectorDoubleToDevice(cu_sdet_vector, metrology.sdet.begin(), met_length));

        cudaSafeCall(hipMalloc((void ** )&cu_fdet_vector, sizeof(*cu_fdet_vector) * met_length));
        cudaSafeCall(detMemcpyVectorDoubleToDevice(cu_fdet_vector, metrology.fdet.begin(), met_length));

        cudaSafeCall(hipMalloc((void ** )&cu_odet_vector, sizeof(*cu_odet_vector) * met_length));
        cudaSafeCall(detMemcpyVectorDoubleToDevice(cu_odet_vector, metrology.odet.begin(), met_length));

        cudaSafeCall(hipMalloc((void ** )&cu_pix0_vector, sizeof(*cu_pix0_vector) * met_length));
        cudaSafeCall(detMemcpyVectorDoubleToDevice(cu_pix0_vector, metrology.pix0.begin(), met_length));

        cudaSafeCall(hipMalloc((void ** )&cu_distance, sizeof(*cu_distance) * metrology.dists.size()));
        cudaSafeCall(detMemcpyVectorDoubleToDevice(cu_distance, metrology.dists.begin(), metrology.dists.size()));

        cudaSafeCall(hipMalloc((void ** )&cu_Xbeam,    sizeof(*cu_Xbeam) * metrology.Xbeam.size()));
        cudaSafeCall(detMemcpyVectorDoubleToDevice(cu_Xbeam,    metrology.Xbeam.begin(), metrology.Xbeam.size()));

        cudaSafeCall(hipMalloc((void ** )&cu_Ybeam,    sizeof(*cu_Ybeam) * metrology.Ybeam.size()));
        cudaSafeCall(detMemcpyVectorDoubleToDevice(cu_Ybeam,    metrology.Ybeam.begin(), metrology.Ybeam.size()));
  }

  void
  gpu_detector::each_image_free(){
    hipSetDevice(h_deviceID);
    cudaSafeCall(hipDeviceSynchronize());
    cudaSafeCall(hipFree(cu_omega_reduction));
    cudaSafeCall(hipFree(cu_max_I_x_reduction));
    cudaSafeCall(hipFree(cu_max_I_y_reduction));
    cudaSafeCall(hipFree(cu_rangemap));
    cudaSafeCall(hipFree(cu_maskimage));
    cudaSafeCall(hipFree(cu_floatimage));
    cudaSafeCall(hipFree(cu_sdet_vector));
    cudaSafeCall(hipFree(cu_fdet_vector));
    cudaSafeCall(hipFree(cu_odet_vector));
    cudaSafeCall(hipFree(cu_pix0_vector));
    cudaSafeCall(hipFree(cu_distance));
    cudaSafeCall(hipFree(cu_Xbeam));
    cudaSafeCall(hipFree(cu_Ybeam));
    cudaSafeCall(hipFree(cu_active_pixel_list));
  }

} // gpu
} // simtbx
