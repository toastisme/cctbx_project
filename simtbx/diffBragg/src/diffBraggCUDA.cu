#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "diffBraggCUDA.h"
#include "diffBragg_gpu_kernel.h"
#include <stdio.h>
//lkalskdlaksdlkalsd

//#define BLOCKSIZE 128
//#define NUMBLOCKS 128
//https://stackoverflow.com/a/14038590/2077270
#define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void error_msg(hipError_t err, char* msg){
    if (err != hipSuccess){
        printf("%s: CUDA error message: %s\n", msg, hipGetErrorString(err));
        exit(err);
    }
}

void diffBragg_loopy(
        int Npix_to_model, std::vector<unsigned int>& panels_fasts_slows,
        image_type& floatimage,
        image_type& d_Umat_images, image_type& d2_Umat_images,
        image_type& d_Bmat_images, image_type& d2_Bmat_images,
        image_type& d_Ncells_images, image_type& d2_Ncells_images,
        image_type& d_fcell_images, image_type& d2_fcell_images,
        image_type& d_eta_images, image_type& d2_eta_images,
        image_type& d_lambda_images, image_type& d2_lambda_images,
        image_type& d_panel_rot_images, image_type& d2_panel_rot_images,
        image_type& d_panel_orig_images, image_type& d2_panel_orig_images,
        image_type& d_sausage_XYZ_scale_images,
        image_type& d_fp_fdp_images,
        const int Nsteps, int _printout_fpixel, int _printout_spixel, bool _printout, CUDAREAL _default_F,
        int oversample, bool _oversample_omega, CUDAREAL subpixel_size, CUDAREAL pixel_size,
        CUDAREAL detector_thickstep, CUDAREAL _detector_thick, CUDAREAL close_distance, CUDAREAL detector_attnlen,
        bool use_lambda_coefficients, CUDAREAL lambda0, CUDAREAL lambda1,
        MAT3& eig_U, MAT3& eig_O, MAT3& eig_B, MAT3& RXYZ,
        std::vector<VEC3,Eigen::aligned_allocator<VEC3> >& dF_vecs,
        std::vector<VEC3,Eigen::aligned_allocator<VEC3> >& dS_vecs,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& UMATS_RXYZ,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& UMATS_RXYZ_prime,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& UMATS_RXYZ_dbl_prime,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& RotMats,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& dRotMats,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& d2RotMats,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& UMATS,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& dB_Mats,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& dB2_Mats,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >&sausages_RXYZ,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& d_sausages_RXYZ,
        std::vector<MAT3,Eigen::aligned_allocator<MAT3> >& sausages_U,
        image_type& sausages_scale, // TODO adjust sausages_scale type
        CUDAREAL* source_X, CUDAREAL* source_Y, CUDAREAL* source_Z, CUDAREAL* source_lambda, CUDAREAL* source_I,
        CUDAREAL kahn_factor,
        CUDAREAL Na, CUDAREAL Nb, CUDAREAL Nc,
        CUDAREAL Nd, CUDAREAL Ne, CUDAREAL Nf,
        CUDAREAL phi0, CUDAREAL phistep,
        VEC3& spindle_vec, VEC3 _polarization_axis,
        int h_range, int k_range, int l_range,
        int h_max, int h_min, int k_max, int k_min, int l_max, int l_min, CUDAREAL dmin,
        CUDAREAL fudge, bool complex_miller, int verbose, bool only_save_omega_kahn,
        bool isotropic_ncells, bool compute_curvatures,
        std::vector<CUDAREAL>& _FhklLinear, std::vector<CUDAREAL>& _Fhkl2Linear,
        std::vector<bool>& refine_Bmat, std::vector<bool>& refine_Ncells, bool refine_Ncells_def, std::vector<bool>& refine_panel_origin,
        std::vector<bool>& refine_panel_rot,
        bool refine_fcell, std::vector<bool>& refine_lambda, bool refine_eta, std::vector<bool>& refine_Umat,
        bool refine_sausages, int num_sausages,
        bool refine_fp_fdp,
        std::vector<CUDAREAL>& fdet_vectors, std::vector<CUDAREAL>& sdet_vectors,
        std::vector<CUDAREAL>& odet_vectors, std::vector<CUDAREAL>& pix0_vectors,
        bool _nopolar, bool _point_pixel, CUDAREAL _fluence, CUDAREAL _r_e_sqr, CUDAREAL _spot_scale,
        int number_of_sources, int device_Id,
        diffBragg_cudaPointers& cp,
        bool update_step_positions, bool update_panels_fasts_slows, bool update_sources, bool update_umats,
        bool update_dB_mats, bool update_rotmats, bool update_Fhkl, bool update_detector, bool update_refine_flags ,
        bool update_panel_deriv_vecs, bool update_sausages_on_device, int detector_thicksteps, int phisteps,
        int Npix_to_allocate, bool no_Nabc_scale,
        std::vector<CUDAREAL>& fpfdp,
        std::vector<CUDAREAL>& fpfdp_derivs,
        std::vector<CUDAREAL>&atom_data){ // diffBragg cuda loopy

    bool ALLOC = !cp.device_is_allocated;

    if (phi0 != 0 || phisteps > 1){
        printf("PHI (goniometer position) not supported in GPU code: phi0=%f phisteps=%d phistep=%f\n", phi0, phisteps, phistep);
        exit(-1);
    }

    int numblocks;
    int blocksize;
    char* diffBragg_blocks = getenv("DIFFBRAGG_NUM_BLOCKS");
    char* diffBragg_threads = getenv("DIFFBRAGG_THREADS_PER_BLOCK");
    if (diffBragg_threads==NULL)
        blocksize=128;
    else
        blocksize=atoi(diffBragg_threads);

    if (diffBragg_blocks==NULL)
        numblocks = (Npix_to_model+blocksize-1)/blocksize;
    else
        numblocks = atoi(diffBragg_blocks);

    int cuda_devices;
    hipGetDeviceCount(&cuda_devices);

    if (num_sausages > 6){
        printf("Too many sausages! Should be less than 6 to run on GPU\n");
        exit(-1);
    }

    error_msg(hipGetLastError(), "after device count");
    if (verbose > 1)
        printf("Found %d CUDA-capable devices\n", cuda_devices);

    //if (device_Id <= cuda_devices)
    gpuErr(hipSetDevice(device_Id));

    double time;
    struct timeval t1, t2, t3 ,t4;
    gettimeofday(&t1, 0);

    if (Npix_to_allocate==-1){
        Npix_to_allocate = Npix_to_model;
    }
    else if (Npix_to_model > Npix_to_allocate){
        printf("Npix to model=%d is greater than the number of pixel requested for allocation (%d)!\n",
            Npix_to_model, Npix_to_allocate);
        exit(-1);
    }


    if (ALLOC){
        if (verbose){
            printf("Will model %d pixels and allocate %d pix\n", Npix_to_model, Npix_to_allocate);
        }
        gpuErr(hipMallocManaged(&cp.cu_source_X, number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_Y, number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_Z, number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_I, number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_lambda, number_of_sources*sizeof(CUDAREAL)));

        gpuErr(hipMallocManaged((void **)&cp.cu_UMATS, UMATS.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_UMATS_RXYZ, UMATS_RXYZ.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_AMATS, UMATS_RXYZ.size()*sausages_U.size()*sizeof(MAT3)));
        if (UMATS_RXYZ_prime.size()>0)
            gpuErr(hipMallocManaged((void **)&cp.cu_UMATS_RXYZ_prime, UMATS_RXYZ_prime.size()*sizeof(MAT3)));
        if (UMATS_RXYZ_dbl_prime.size()>0)
            gpuErr(hipMallocManaged((void **)&cp.cu_UMATS_RXYZ_dbl_prime, UMATS_RXYZ_dbl_prime.size()*sizeof(MAT3)));

        gpuErr(hipMallocManaged((void **)&cp.cu_dB_Mats, dB_Mats.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_dB2_Mats, dB2_Mats.size()*sizeof(MAT3)));

        gpuErr(hipMallocManaged((void **)&cp.cu_RotMats, RotMats.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_dRotMats, dRotMats.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_d2RotMats, d2RotMats.size()*sizeof(MAT3)));

        gpuErr(hipMallocManaged(&cp.cu_fdet_vectors, fdet_vectors.size()*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_sdet_vectors, fdet_vectors.size()*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_odet_vectors, fdet_vectors.size()*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_pix0_vectors, fdet_vectors.size()*sizeof(CUDAREAL)));

        if (fpfdp.size() > 0){
            gpuErr(hipMallocManaged(&cp.cu_fpfdp, fpfdp.size()*sizeof(CUDAREAL)));
            gpuErr(hipMallocManaged(&cp.cu_atom_data, atom_data.size()*sizeof(CUDAREAL)));
        }
        if(fpfdp_derivs.size() > 0)
            gpuErr(hipMallocManaged(&cp.cu_fpfdp_derivs, fpfdp_derivs.size()*sizeof(CUDAREAL)));

        gpuErr(hipMallocManaged(&cp.cu_refine_Bmat, 6*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_Umat, 3*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_Ncells, 3*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_panel_origin, 3*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_panel_rot, 3*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_lambda, 2*sizeof(bool)));

        gpuErr(hipMallocManaged(&cp.cu_Fhkl, _FhklLinear.size()*sizeof(CUDAREAL)));
        if (complex_miller)
            gpuErr(hipMallocManaged(&cp.cu_Fhkl2, _FhklLinear.size()*sizeof(CUDAREAL)));

        gpuErr(hipMallocManaged((void **)&cp.cu_dF_vecs, dF_vecs.size()*sizeof(VEC3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_dS_vecs, dF_vecs.size()*sizeof(VEC3)));

        gpuErr(hipMallocManaged( (void**)&cp.cu_sausages_RXYZ, sausages_RXYZ.size()*sizeof(MAT3) ));
        gpuErr(hipMallocManaged( (void**)&cp.cu_d_sausages_RXYZ, d_sausages_RXYZ.size()*sizeof(MAT3) ));
        gpuErr(hipMallocManaged( (void**)&cp.cu_sausages_U, sausages_U.size()*sizeof(MAT3) ));
        gpuErr(hipMallocManaged( &cp.cu_sausages_scale, sausages_scale.size()*sizeof(CUDAREAL) ));

        //gettimeofday(&t3, 0));
        gpuErr(hipMallocManaged(&cp.cu_floatimage, Npix_to_allocate*sizeof(CUDAREAL) ));
        gpuErr(hipMallocManaged(&cp.cu_d_fcell_images, Npix_to_allocate*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d_eta_images, Npix_to_allocate*3*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d2_eta_images, Npix_to_allocate*3*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d_Umat_images, Npix_to_allocate*3*sizeof(CUDAREAL) ));
        gpuErr(hipMallocManaged(&cp.cu_d_Ncells_images, Npix_to_allocate*6*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d_panel_rot_images, Npix_to_allocate*3*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d_panel_orig_images, Npix_to_allocate*3*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d_lambda_images, Npix_to_allocate*2*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d_Bmat_images, Npix_to_allocate*6*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d_sausage_XYZ_scale_images, num_sausages*Npix_to_allocate*4*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d_fp_fdp_images, Npix_to_allocate*2*sizeof(CUDAREAL)));

        // allocate curvatures
        //gpuErr(hipMallocManaged(&cp.cu_d_eta_images, Npix_to_allocate*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d2_Umat_images, Npix_to_allocate*3*sizeof(CUDAREAL) ));
        gpuErr(hipMallocManaged(&cp.cu_d2_Ncells_images, Npix_to_allocate*6*sizeof(CUDAREAL)));
        //gpuErr(hipMallocManaged(&cp.cu_d_panel_rot_images, Npix_to_allocate*3*sizeof(CUDAREAL)));
        //gpuErr(hipMallocManaged(&cp.cu_d_panel_orig_images, Npix_to_allocate*3*sizeof(CUDAREAL)));
        //gpuErr(hipMallocManaged(&cp.cu_d_lambda_images, Npix_to_allocate*2*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_d2_Bmat_images, Npix_to_allocate*6*sizeof(CUDAREAL)));
        //gpuErr(hipMallocManaged(&cp.cu_d_sausage_XYZ_scale_images, num_sausages*Npix_to_allocate*4*sizeof(CUDAREAL)));

        //gettimeofday(&t4, 0);
        //time = (1000000.0*(t4.tv_sec-t3.tv_sec) + t4.tv_usec-t3.tv_usec)/1000.0;
        //printf("TIME SPENT ALLOCATING (IMAGES ONLY):  %3.10f ms \n", time);
        gpuErr(hipMallocManaged(&cp.cu_panels_fasts_slows, Npix_to_allocate*3*sizeof(panels_fasts_slows[0])));
    } // END ALLOC

    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if(verbose>1)
        printf("TIME SPENT ALLOCATING (TOTAL):  %3.10f ms \n", time);

    //ALLOC = false;
//  BEGIN COPYING DATA
    gettimeofday(&t1, 0);
    bool FORCE_COPY=true;

//  END step position
    int kaladin_stormblessed = 777;


//  BEGIN sources
    if (update_sources || ALLOC || FORCE_COPY){
        for (int i=0; i< number_of_sources; i++){
            VEC3 incident(source_X[i], source_Y[i], source_Z[i]);
            incident /= incident.norm();
            cp.cu_source_X[i] = incident[0];
            cp.cu_source_Y[i] = incident[1];
            cp.cu_source_Z[i] = incident[2];
            cp.cu_source_I[i] = source_I[i];
            cp.cu_source_lambda[i] = source_lambda[i];
        }
        if(verbose>1 )
          printf("H2D sources\n");
    }
//  END sources


//  UMATS
    if (update_umats || ALLOC||FORCE_COPY){
        for (int i=0; i< UMATS.size(); i++)
            cp.cu_UMATS[i] = UMATS[i];
        //int idx=0;
        //for (int i=0; i < UMATS_RXYZ.size(); i++){
        //    for (auto elem: UMATS_RXYZ[i].reshaped()){
        //        cp.cu_UMATS_RXYZ[idx] = elem;
        //        idx ++;
        //    }
        //}
        for (int i=0; i < UMATS_RXYZ.size(); i++)
            cp.cu_UMATS_RXYZ[i] = UMATS_RXYZ[i];
        for (int i=0; i < UMATS_RXYZ_prime.size(); i++)
            cp.cu_UMATS_RXYZ_prime[i] = UMATS_RXYZ_prime[i];
        for (int i=0; i < UMATS_RXYZ_dbl_prime.size(); i++)
            cp.cu_UMATS_RXYZ_dbl_prime[i] = UMATS_RXYZ_dbl_prime[i];
        if(verbose>1)
            printf("H2D Done copying Umats\n") ;
    }
//  END UMATS


    if (update_umats || update_sausages_on_device|| ALLOC||FORCE_COPY){
        MAT3 Amat_init = eig_U*eig_B*1e10*(eig_O.transpose());
        for (int i_sausage=0; i_sausage< sausages_U.size(); i_sausage++){
            for(int i_mos =0; i_mos< UMATS_RXYZ.size(); i_mos++){
                int idx = UMATS_RXYZ.size()*i_sausage + i_mos;
                cp.cu_AMATS[idx] = (UMATS_RXYZ[i_mos]*sausages_U[i_sausage]*Amat_init).transpose();
            }
        }
        if(verbose>1)
            printf("H2D Done copying Amats\n") ;
    }


//  BMATS
    if(update_dB_mats || ALLOC || FORCE_COPY){
        for (int i=0; i< dB_Mats.size(); i++)
            cp.cu_dB_Mats[i] = dB_Mats[i];
        for (int i=0; i< dB2_Mats.size(); i++)
            cp.cu_dB2_Mats[i] = dB2_Mats[i];
        if(verbose>1)
            printf("H2D Done copying dB_Mats\n") ;
    }
//  END BMATS


//  ROT MATS
    if(update_rotmats || ALLOC || FORCE_COPY){
        for (int i=0; i<RotMats.size(); i++)
            cp.cu_RotMats[i] = RotMats[i];
        for (int i=0; i<dRotMats.size(); i++)
            cp.cu_dRotMats[i] = dRotMats[i];
        for (int i=0; i<d2RotMats.size(); i++)
            cp.cu_d2RotMats[i] = d2RotMats[i];
        if (verbose>1)
          printf("H2D Done copying rotmats\n");
    }
//  END ROT MATS

//  sausages
    if(update_sausages_on_device || ALLOC || FORCE_COPY){
        for (int i=0; i<sausages_RXYZ.size(); i++)
            cp.cu_sausages_RXYZ[i] = sausages_RXYZ[i];
        for (int i=0; i<sausages_U.size(); i++)
            cp.cu_sausages_U[i] = sausages_U[i];
        for (int i=0; i<d_sausages_RXYZ.size(); i++)
            cp.cu_d_sausages_RXYZ[i] = d_sausages_RXYZ[i];
        for (int i=0; i< sausages_scale.size(); i++)
            cp.cu_sausages_scale[i] = sausages_scale[i];
        if (verbose>1)
          printf("H2D Done copying sausages\n");
    }
//  END ROT MATS


//  DETECTOR VECTORS
    if (update_detector || ALLOC || FORCE_COPY){
        for (int i=0; i<fdet_vectors.size(); i++){
            cp.cu_fdet_vectors[i] = fdet_vectors[i];
            cp.cu_sdet_vectors[i] = sdet_vectors[i];
            cp.cu_odet_vectors[i] = odet_vectors[i];
            cp.cu_pix0_vectors[i] = pix0_vectors[i];
        }
        if (verbose>1)
          printf("H2D Done copying detector vectors\n");
    }
//  END  DETECTOR VECTORS

    if ( ALLOC || FORCE_COPY){
      for (int i=0; i< atom_data.size(); i++){
        cp.cu_atom_data[i] = atom_data[i];
      }
      if (verbose>1)
        printf("H2D Done copying atom data\n");
      for(int i=0; i< fpfdp.size(); i++){
        cp.cu_fpfdp[i] = fpfdp[i];
      }
      for(int i=0; i< fpfdp_derivs.size(); i++){
        cp.cu_fpfdp_derivs[i] = fpfdp_derivs[i];
      }
      if (verbose>1)
        printf("H2D Done copying fprime and fdblprime\n");
    }


//  BEGIN REFINEMENT FLAGS
    if (update_refine_flags || ALLOC || FORCE_COPY){
        for (int i=0; i<3; i++){
            cp.cu_refine_Umat[i] = refine_Umat[i];
            cp.cu_refine_Ncells[i] = refine_Ncells[i];
            cp.cu_refine_panel_origin[i] = refine_panel_origin[i];
            cp.cu_refine_panel_rot[i] = refine_panel_rot[i];
        }
        for(int i=0; i<2; i++)
            cp.cu_refine_lambda[i] = refine_lambda[i];
        for(int i=0; i<6; i++)
            cp.cu_refine_Bmat[i] = refine_Bmat[i];
        if (verbose>1)
          printf("H2D Done copying refinement flags\n");
    }
//  END REFINEMENT FLAGS


//  BEGIN Fhkl
    if (update_Fhkl || ALLOC || FORCE_COPY){
        for(int i=0; i < _FhklLinear.size(); i++){
          cp.cu_Fhkl[i] = _FhklLinear[i];
          if (complex_miller)
              cp.cu_Fhkl2[i] = _Fhkl2Linear[i];
        }
        if (verbose>1)
            printf("H2D Done copying step Fhkl\n");
    }
//  END Fhkl

//  BEGIN panel derivative vecs
    if(update_panel_deriv_vecs || ALLOC || FORCE_COPY){
        for (int i=0; i<dF_vecs.size(); i++){
            cp.cu_dF_vecs[i] = dF_vecs[i];
            cp.cu_dS_vecs[i] = dS_vecs[i];
        }
        if (verbose>1)
            printf("H2D Done copying step panel derivative vectors\n");
    }
//  END panel derivative vecs

//  BEGIN panels fasts slows
    if (update_panels_fasts_slows || ALLOC || FORCE_COPY){
        for (int i=0; i< panels_fasts_slows.size(); i++)
            cp.cu_panels_fasts_slows[i] = panels_fasts_slows[i];
        if (verbose>1)
            printf("H2D Done copying panels_fasts_slows\n");
    }
//  END panels fasts slows


    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if(verbose>1)
        printf("TIME SPENT COPYING DATA HOST->DEV:  %3.10f ms \n", time);

    cp.device_is_allocated = true;
    error_msg(hipGetLastError(), "after copy to device");

    gettimeofday(&t1, 0);

    int Npanels = fdet_vectors.size()/3;
    int num_atoms = atom_data.size()/5;
    //int sm_size = number_of_sources*5*sizeof(CUDAREAL);
    //gpu_sum_over_steps<<<numblocks, blocksize, sm_size >>>(
    bool aniso_eta = UMATS_RXYZ.size() != UMATS_RXYZ_prime.size();
    gpu_sum_over_steps<<<numblocks, blocksize>>>(
        Npix_to_model, cp.cu_panels_fasts_slows,
        cp.cu_floatimage,
        cp.cu_d_Umat_images, cp.cu_d2_Umat_images,
        cp.cu_d_Bmat_images, cp.cu_d2_Bmat_images,
        cp.cu_d_Ncells_images, cp.cu_d2_Ncells_images,
        cp.cu_d_fcell_images, cp.cu_d2_fcell_images,
        cp.cu_d_eta_images, cp.cu_d2_eta_images,
        cp.cu_d_lambda_images, cp.cu_d2_lambda_images,
        cp.cu_d_panel_rot_images, cp.cu_d2_panel_rot_images,
        cp.cu_d_panel_orig_images, cp.cu_d2_panel_orig_images,
        cp.cu_d_sausage_XYZ_scale_images,
        cp.cu_d_fp_fdp_images,
        Nsteps, _printout_fpixel, _printout_spixel, _printout, _default_F,
        oversample,  _oversample_omega, subpixel_size, pixel_size,
        detector_thickstep, _detector_thick, close_distance, detector_attnlen,
        detector_thicksteps, number_of_sources, phisteps, UMATS.size(),
        use_lambda_coefficients, lambda0, lambda1,
        eig_U, eig_O, eig_B, RXYZ,
        cp.cu_dF_vecs,
        cp.cu_dS_vecs,
        cp.cu_UMATS_RXYZ,
        cp.cu_UMATS_RXYZ_prime,
        cp.cu_UMATS_RXYZ_dbl_prime,
        cp.cu_RotMats,
        cp.cu_dRotMats,
        cp.cu_d2RotMats,
        cp.cu_UMATS,
        cp.cu_dB_Mats,
        cp.cu_dB2_Mats,
        cp.cu_AMATS,
        cp.cu_sausages_RXYZ, cp.cu_d_sausages_RXYZ, cp.cu_sausages_U, cp.cu_sausages_scale,
        cp.cu_source_X, cp.cu_source_Y, cp.cu_source_Z, cp.cu_source_lambda, cp.cu_source_I,
        kahn_factor,
        Na, Nb, Nc,
        Nd, Ne, Nf,
        phi0, phistep,
        spindle_vec, _polarization_axis,
        h_range, k_range, l_range,
        h_max, h_min, k_max, k_min, l_max, l_min, dmin,
        fudge, complex_miller, verbose, only_save_omega_kahn,
        isotropic_ncells, compute_curvatures,
        cp.cu_Fhkl, cp.cu_Fhkl2,
        cp.cu_refine_Bmat, cp.cu_refine_Ncells, refine_Ncells_def, cp.cu_refine_panel_origin, cp.cu_refine_panel_rot,
        refine_fcell, cp.cu_refine_lambda, refine_eta, cp.cu_refine_Umat,
        refine_sausages, num_sausages,
        cp.cu_fdet_vectors, cp.cu_sdet_vectors,
        cp.cu_odet_vectors, cp.cu_pix0_vectors,
        _nopolar, _point_pixel, _fluence, _r_e_sqr, _spot_scale, Npanels, aniso_eta, no_Nabc_scale,
        cp.cu_fpfdp,  cp.cu_fpfdp_derivs, cp.cu_atom_data, num_atoms,
        refine_fp_fdp);

    error_msg(hipGetLastError(), "after kernel call");

    hipDeviceSynchronize();
    error_msg(hipGetLastError(), "after kernel completion");

    if(verbose>1)
        printf("KERNEL_COMPLETE gpu_sum_over_steps\n");
    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if(verbose>1)
        printf("TIME SPENT(KERNEL):  %3.10f ms \n", time);

    gettimeofday(&t1, 0);
//  COPY BACK FROM DEVICE
    for (int i=0; i< Npix_to_model; i++){
        floatimage[i] = cp.cu_floatimage[i];
        d_fcell_images[i] = cp.cu_d_fcell_images[i];
    }
    for (int i=0; i<3*Npix_to_model; i++){
        d_Umat_images[i] = cp.cu_d_Umat_images[i];
        d2_Umat_images[i] = cp.cu_d2_Umat_images[i];
        d_panel_rot_images[i] = cp.cu_d_panel_rot_images[i];
        d_panel_orig_images[i] = cp.cu_d_panel_orig_images[i];
        d_eta_images[i] = cp.cu_d_eta_images[i];
        d2_eta_images[i] = cp.cu_d2_eta_images[i];
    }

    for(int i=0; i<6*Npix_to_model; i++){
        d_Ncells_images[i] = cp.cu_d_Ncells_images[i];
        d2_Ncells_images[i] = cp.cu_d2_Ncells_images[i];
        d_Bmat_images[i] = cp.cu_d_Bmat_images[i];
        d2_Bmat_images[i] = cp.cu_d2_Bmat_images[i];
    }
    for(int i=0; i<2*Npix_to_model; i++)
        d_lambda_images[i] = cp.cu_d_lambda_images[i];

    for (int i=0; i< num_sausages*4*Npix_to_model; i++)
        d_sausage_XYZ_scale_images[i] = cp.cu_d_sausage_XYZ_scale_images[i];

    for (int i=0; i< 2*Npix_to_model; i++)
        d_fp_fdp_images[i] = cp.cu_d_fp_fdp_images[i];

    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if(verbose>1)
        printf("TIME SPENT COPYING BACK :  %3.10f ms \n", time);
    error_msg(hipGetLastError(), "After copy to host");
}


void freedom(diffBragg_cudaPointers& cp){

    if (cp.device_is_allocated){
        gpuErr(hipFree( cp.cu_floatimage));
        gpuErr(hipFree( cp.cu_d_Umat_images));
        gpuErr(hipFree( cp.cu_d_Bmat_images));
        gpuErr(hipFree( cp.cu_d_Ncells_images));
        gpuErr(hipFree( cp.cu_d2_Umat_images));
        gpuErr(hipFree( cp.cu_d2_Bmat_images));
        gpuErr(hipFree( cp.cu_d2_Ncells_images));
        gpuErr(hipFree( cp.cu_d_eta_images));
        gpuErr(hipFree( cp.cu_d2_eta_images));
        gpuErr(hipFree( cp.cu_d_fcell_images));
        gpuErr(hipFree( cp.cu_d_lambda_images));
        gpuErr(hipFree( cp.cu_d_panel_rot_images));
        gpuErr(hipFree( cp.cu_d_panel_orig_images));
        gpuErr(hipFree( cp.cu_d_sausage_XYZ_scale_images));
        gpuErr(hipFree( cp.cu_d_fp_fdp_images));

        gpuErr(hipFree(cp.cu_Fhkl));
        if (cp.cu_Fhkl2 != NULL)
            gpuErr(hipFree(cp.cu_Fhkl2));

        gpuErr(hipFree(cp.cu_fdet_vectors));
        gpuErr(hipFree(cp.cu_sdet_vectors));
        gpuErr(hipFree(cp.cu_odet_vectors));
        gpuErr(hipFree(cp.cu_pix0_vectors));
        gpuErr(hipFree(cp.cu_atom_data));
        gpuErr(hipFree(cp.cu_fpfdp));
        gpuErr(hipFree(cp.cu_fpfdp_derivs));

        gpuErr(hipFree(cp.cu_source_X));
        gpuErr(hipFree(cp.cu_source_Y));
        gpuErr(hipFree(cp.cu_source_Z));
        gpuErr(hipFree(cp.cu_source_I));
        gpuErr(hipFree(cp.cu_source_lambda));

        gpuErr(hipFree(cp.cu_UMATS));
        gpuErr(hipFree(cp.cu_UMATS_RXYZ));
        gpuErr(hipFree(cp.cu_AMATS));
        if(cp.cu_UMATS_RXYZ_prime != NULL)
            gpuErr(hipFree(cp.cu_UMATS_RXYZ_prime));
        if(cp.cu_UMATS_RXYZ_dbl_prime != NULL)
            gpuErr(hipFree(cp.cu_UMATS_RXYZ_dbl_prime));
        gpuErr(hipFree(cp.cu_RotMats));
        gpuErr(hipFree(cp.cu_dRotMats));
        gpuErr(hipFree(cp.cu_d2RotMats));
        gpuErr(hipFree(cp.cu_dB_Mats));
        gpuErr(hipFree(cp.cu_dB2_Mats));
        gpuErr(hipFree(cp.cu_sausages_RXYZ));
        gpuErr(hipFree(cp.cu_d_sausages_RXYZ));
        gpuErr(hipFree(cp.cu_sausages_U));
        gpuErr(hipFree(cp.cu_sausages_scale));

        gpuErr(hipFree(cp.cu_dF_vecs));
        gpuErr(hipFree(cp.cu_dS_vecs));

        gpuErr(hipFree(cp.cu_refine_Bmat));
        gpuErr(hipFree(cp.cu_refine_Umat));
        gpuErr(hipFree(cp.cu_refine_Ncells));
        gpuErr(hipFree(cp.cu_refine_lambda));
        gpuErr(hipFree(cp.cu_refine_panel_origin));
        gpuErr(hipFree(cp.cu_refine_panel_rot));

        gpuErr(hipFree(cp.cu_panels_fasts_slows));

        cp.device_is_allocated = false;
    }
}



// Kernel function to add the elements of two arrays
__global__
void phat_add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int phat_main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory  accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  phat_add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
